#include "kernels.h"

__host__ void set_block_header(struct worker_t *worker, uint32_t threadIndex, nimiq_block_header *block_header)
{
    initial_seed inseed;
    inseed.lanes = 1;
    inseed.hash_len = ARGON2_HASH_LENGTH;
    inseed.memory_cost = NIMIQ_ARGON2_COST;
    inseed.iterations = 1;
    inseed.version = 0x13;
    inseed.type = 0;
    inseed.header_len = sizeof(nimiq_block_header);
    memcpy(&inseed.header, block_header, sizeof(nimiq_block_header));
    inseed.salt_len = NIMIQ_ARGON2_SALT_LEN;
    memcpy(&inseed.salt, NIMIQ_ARGON2_SALT, NIMIQ_ARGON2_SALT_LEN);
    inseed.secret_len = 0;
    inseed.extra_len = 0;
    memset(&inseed.padding, 0, sizeof(inseed.padding));
  
    hipMemcpyAsync(worker->inseed[threadIndex], &inseed, sizeof(initial_seed), hipMemcpyHostToDevice);
    hipMemsetAsync(worker->nonce[threadIndex], 0, sizeof(uint32_t)); // zero nonce
}

__host__ uint32_t mine_nonces(struct worker_t *worker, uint32_t threadIndex, uint32_t start_nonce, uint32_t share_compact)
{
    init_memory<<<worker->init_memory_blocks, worker->init_memory_threads>>>(worker->memory[threadIndex], worker->inseed[threadIndex], start_nonce);
    argon2<<<worker->argon2_blocks, worker->argon2_threads, worker->cacheSize * ARGON2_BLOCK_SIZE>>>(worker->memory[threadIndex], worker->cacheSize, worker->memoryTradeoff);
    get_nonce<<<worker->get_nonce_blocks, worker->get_nonce_threads>>>(worker->memory[threadIndex], start_nonce, share_compact, worker->nonce[threadIndex]);

    hipStreamSynchronize(0);

    uint32_t nonce;
    hipMemcpy(&nonce, worker->nonce[threadIndex], sizeof(uint32_t), hipMemcpyDeviceToHost);

    if (nonce > 0)
    {
        hipMemsetAsync(worker->nonce[threadIndex], 0, sizeof(uint32_t)); // zero nonce
    }
    return nonce;
}
