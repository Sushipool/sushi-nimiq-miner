#include "hip/hip_runtime.h"
/*
MIT License

Copyright (c) 2016 Ondrej Mosnáček

Permission is hereby granted, free of charge, to any person obtaining a copy
of this software and associated documentation files (the "Software"), to deal
in the Software without restriction, including without limitation the rights
to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
copies of the Software, and to permit persons to whom the Software is
furnished to do so, subject to the following conditions:

The above copyright notice and this permission notice shall be included in all
copies or substantial portions of the Software.

THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
SOFTWARE.
*/

/*
* Argon2d
* Simplified version of https://gitlab.com/omos/argon2-gpu
*/

#include "kernels.h"

__device__ uint64_t u64_build(uint32_t hi, uint32_t lo)
{
    return ((uint64_t)hi << 32) | (uint64_t)lo;
}

__device__ uint32_t u64_lo(uint64_t x)
{
    return (uint32_t)x;
}

__device__ uint32_t u64_hi(uint64_t x)
{
    return (uint32_t)(x >> 32);
}

__device__ uint64_t u64_shuffle(uint64_t v, uint32_t thread)
{
    uint32_t lo = u64_lo(v);
    uint32_t hi = u64_hi(v);
    lo = __shfl_sync(0xFFFFFFFF, lo, thread);
    hi = __shfl_sync(0xFFFFFFFF, hi, thread);
    return u64_build(hi, lo);
}

struct block_th
{
    uint64_t a, b, c, d;
};

__device__ uint64_t cmpeq_mask(uint32_t test, uint32_t ref)
{
    uint32_t x = -(uint32_t)(test == ref);
    return u64_build(x, x);
}

__device__ uint64_t block_th_get(const struct block_th *b, uint32_t idx)
{
    uint64_t res = 0;
    res ^= cmpeq_mask(idx, 0) & b->a;
    res ^= cmpeq_mask(idx, 1) & b->b;
    res ^= cmpeq_mask(idx, 2) & b->c;
    res ^= cmpeq_mask(idx, 3) & b->d;
    return res;
}

__device__ void block_th_set(struct block_th *b, uint32_t idx, uint64_t v)
{
    b->a ^= cmpeq_mask(idx, 0) & (v ^ b->a);
    b->b ^= cmpeq_mask(idx, 1) & (v ^ b->b);
    b->c ^= cmpeq_mask(idx, 2) & (v ^ b->c);
    b->d ^= cmpeq_mask(idx, 3) & (v ^ b->d);
}

__device__ void move_block(struct block_th *dst, const struct block_th *src)
{
    *dst = *src;
}

__device__ void xor_block(struct block_th *dst, const struct block_th *src)
{
    dst->a ^= src->a;
    dst->b ^= src->b;
    dst->c ^= src->c;
    dst->d ^= src->d;
}

__device__ void load_block(struct block_th *dst, const struct block_g *src, uint32_t thread)
{
    dst->a = src->data[0 * THREADS_PER_LANE + thread];
    dst->b = src->data[1 * THREADS_PER_LANE + thread];
    dst->c = src->data[2 * THREADS_PER_LANE + thread];
    dst->d = src->data[3 * THREADS_PER_LANE + thread];
}

__device__ void load_block_xor(struct block_th *dst, const struct block_g *src, uint32_t thread)
{
    dst->a ^= src->data[0 * THREADS_PER_LANE + thread];
    dst->b ^= src->data[1 * THREADS_PER_LANE + thread];
    dst->c ^= src->data[2 * THREADS_PER_LANE + thread];
    dst->d ^= src->data[3 * THREADS_PER_LANE + thread];
}

__device__ void store_block(struct block_g *dst, const struct block_th *src, uint32_t thread)
{
    dst->data[0 * THREADS_PER_LANE + thread] = src->a;
    dst->data[1 * THREADS_PER_LANE + thread] = src->b;
    dst->data[2 * THREADS_PER_LANE + thread] = src->c;
    dst->data[3 * THREADS_PER_LANE + thread] = src->d;
}

__device__ uint64_t rotr64(uint64_t x, uint32_t n)
{
    return (x >> n) | (x << (64 - n));
}

__device__ uint64_t f(uint64_t x, uint64_t y)
{
    uint32_t xlo = u64_lo(x);
    uint32_t ylo = u64_lo(y);
    return x + y + 2 * u64_build(__umulhi(xlo, ylo), xlo * ylo);
}

__device__ void g(struct block_th *block)
{
    uint64_t a, b, c, d;
    a = block->a;
    b = block->b;
    c = block->c;
    d = block->d;

    a = f(a, b);
    d = rotr64(d ^ a, 32);
    c = f(c, d);
    b = rotr64(b ^ c, 24);
    a = f(a, b);
    d = rotr64(d ^ a, 16);
    c = f(c, d);
    b = rotr64(b ^ c, 63);

    block->a = a;
    block->b = b;
    block->c = c;
    block->d = d;
}

template<class shuffle>
__device__ void apply_shuffle(struct block_th *block, uint32_t thread)
{
    for (uint32_t i = 0; i < QWORDS_PER_THREAD; i++) {
        uint32_t src_thr = shuffle::apply(thread, i);

        uint64_t v = block_th_get(block, i);
        v = u64_shuffle(v, src_thr);
        block_th_set(block, i, v);
    }
}

__device__ void transpose(struct block_th *block, uint32_t thread)
{
    uint32_t thread_group = (thread & 0x0C) >> 2;
    for (uint32_t i = 1; i < QWORDS_PER_THREAD; i++) {
        uint32_t thr = (i << 2) ^ thread;
        uint32_t idx = thread_group ^ i;

        uint64_t v = block_th_get(block, idx);
        v = u64_shuffle(v, thr);
        block_th_set(block, idx, v);
    }
}

struct shift1_shuffle {
    __device__ static uint32_t apply(uint32_t thread, uint32_t idx)
    {
        return (thread & 0x1c) | ((thread + idx) & 0x3);
    }
};

struct unshift1_shuffle {
    __device__ static uint32_t apply(uint32_t thread, uint32_t idx)
    {
        idx = (QWORDS_PER_THREAD - idx) % QWORDS_PER_THREAD;

        return (thread & 0x1c) | ((thread + idx) & 0x3);
    }
};

struct shift2_shuffle {
    __device__ static uint32_t apply(uint32_t thread, uint32_t idx)
    {
        uint32_t lo = (thread & 0x1) | ((thread & 0x10) >> 3);
        lo = (lo + idx) & 0x3;
        return ((lo & 0x2) << 3) | (thread & 0xe) | (lo & 0x1);
    }
};

struct unshift2_shuffle {
    __device__ static uint32_t apply(uint32_t thread, uint32_t idx)
    {
        idx = (QWORDS_PER_THREAD - idx) % QWORDS_PER_THREAD;

        uint32_t lo = (thread & 0x1) | ((thread & 0x10) >> 3);
        lo = (lo + idx) & 0x3;
        return ((lo & 0x2) << 3) | (thread & 0xe) | (lo & 0x1);
    }
};

__device__ void shuffle_block(struct block_th *block, uint32_t thread)
{
    transpose(block, thread);

    g(block);

    apply_shuffle<shift1_shuffle>(block, thread);

    g(block);

    apply_shuffle<unshift1_shuffle>(block, thread);
    transpose(block, thread);

    g(block);

    apply_shuffle<shift2_shuffle>(block, thread);

    g(block);

    apply_shuffle<unshift2_shuffle>(block, thread);
}

__device__ uint32_t compute_ref_index(struct block_th *prev, uint32_t curr_index)
{
    uint64_t v = u64_shuffle(prev->a, 0);
    uint32_t ref_index = u64_lo(v);

    uint32_t ref_area_size = curr_index - 1;
    ref_index = __umulhi(ref_index, ref_index);
    ref_index = ref_area_size - 1 - __umulhi(ref_area_size, ref_index);
    return ref_index;
}

__device__ void load_block(struct block_th *dst,
                           const struct block_g *memory,
                           const struct block_g *cache, uint32_t cacheSize,
                           uint32_t index, uint32_t thread)
{
    if (index < 2 + cacheSize && index >= 2)
    {
        load_block(dst, cache + index - 2, thread);
    }
    else
    {
        load_block(dst, memory + index, thread);
    }
}

__device__ void load_block_xor(struct block_th *dst,
                               const struct block_g *memory,
                               const struct block_g *cache, uint32_t cacheSize,
                               uint32_t index, uint32_t thread)
{
    if (index < 2 + cacheSize && index >= 2)
    {
        load_block_xor(dst, cache + index - 2, thread);
    }
    else
    {
        load_block_xor(dst, memory + index, thread);
    }
}

__device__ void store_block(struct block_g *memory,
                            struct block_g *cache, uint32_t cacheSize,
                            const struct block_th *src,
                            uint32_t index, uint32_t thread)
{
    if (index < 2 + cacheSize && index >= 2)
    {
        store_block(cache + index - 2, src, thread);
    }
    else
    {
        store_block(memory + index, src, thread);
    }
}

__device__ void get_ref_index(uint32_t *ref_index, bool *is_stored, const uint16_t *ref_indexes, uint32_t index)
{
    uint16_t ri = ref_indexes[index];
    *ref_index = (ri & 0x7FFF);
    *is_stored = (bool) (ri & 0x8000);
}

__device__ void set_ref_index(uint16_t *ref_indexes, uint32_t index, uint32_t ref_index, bool is_stored, uint32_t thread)
{
    if (thread == 0)
    {
        ref_indexes[index] = (is_stored ? 0x8000 : 0) | ref_index;
    }
    __syncwarp();
}

__device__ void compute_block_xor(struct block_th *dst,
                                const struct block_g *memory,
                                const struct block_g *cache, uint32_t cacheSize,
                                uint32_t index, uint32_t ref_index, uint32_t thread)
{
    struct block_th prev, tmp;

    load_block(&prev, memory, cache, cacheSize, index - 1, thread);
    load_block_xor(&prev, memory, cache, cacheSize, ref_index, thread);

    move_block(&tmp, &prev);
    shuffle_block(&prev, thread);
    xor_block(&prev, &tmp);

    xor_block(dst, &prev);
}

__device__ void argon2_step(struct block_g *memory, struct block_g *cache, uint32_t cacheSize,
                            uint16_t *ref_indexes, uint32_t memoryTradeoff,
                            uint32_t curr_index, struct block_th *prev, bool *is_prev_stored, uint32_t thread)
{
    struct block_th tmp;
    bool is_ref_stored = true;
    bool is_curr_stored = true;

    uint32_t ref_index = compute_ref_index(prev, curr_index);

    if (curr_index >= memoryTradeoff)
    {
        if (ref_index >= memoryTradeoff && ref_index >= 2)
        {
            // what was the ref block of the current ref block?
            uint32_t ref_ref_index;
            get_ref_index(&ref_ref_index, &is_ref_stored, ref_indexes, ref_index);
            if (!is_ref_stored)
            {
                compute_block_xor(prev, memory, cache, cacheSize, ref_index, ref_ref_index, thread);
            }
        }
        is_curr_stored = !(*is_prev_stored && is_ref_stored) || (curr_index == MEMORY_COST - 1);

        set_ref_index(ref_indexes, curr_index, ref_index, is_curr_stored, thread);
    }

    // load if it was not computed before 
    if (is_ref_stored)
    {
        load_block_xor(prev, memory, cache, cacheSize, ref_index, thread);
    }

    move_block(&tmp, prev);
    shuffle_block(prev, thread);
    xor_block(prev, &tmp);

    if (is_curr_stored)
    {
        store_block(memory, cache, cacheSize, prev, curr_index, thread);
    }
    *is_prev_stored = is_curr_stored;
}

__global__ void argon2(struct block_g *memory, uint32_t cacheSize, uint32_t memoryTradeoff)
{
    extern __shared__ struct block_g cache[];
    // ref_index of the current block, msb = 1 if current block is stored to global mem
    __shared__ uint16_t ref_indexes[MEMORY_COST];

    uint32_t job_id = blockIdx.y;
    uint32_t thread = threadIdx.x;

    /* select job's memory region: */
    memory += (size_t)job_id * MEMORY_COST;

    struct block_th prev;
    bool is_prev_stored = true;

    load_block(&prev, memory + 1, thread);

    for (uint32_t curr_index = 2; curr_index < MEMORY_COST; curr_index++)
    {
        argon2_step(memory, cache, cacheSize, ref_indexes, memoryTradeoff, curr_index, &prev, &is_prev_stored, thread);
    }
}
